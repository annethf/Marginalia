#include "hip/hip_runtime.h"
/**
	һ���̼߳��������Ĳ��ƽ��
	ѭ��1536��
**/
#include "hip/hip_runtime.h"
#include ""
#include "book.h"
#include <iostream>
#include <math.h>
#include <time.h>

using namespace std;

#define THREAD_NUM 1024
#define DATA_SIZE 1572864

void genVector(int *a)
{
	for(int i = 0; i < DATA_SIZE; i++)
		a[i] = rand() % 10;
}

float dis(int *a, int *b)
{
	int sum = 0;
	float d = 0.0;
	for(int i = 0; i < DATA_SIZE; i++)
	{
		int temp = a[i] - b[i];
		sum += temp * temp;
	}
	d = sqrt((float)sum);
	return d;
}

//��������������Ӧ�����Ĳ��ƽ��
__global__ void vectorDistance(int *a, int *b, int *sum)
{
	unsigned int tid = threadIdx.x;
	if(tid < THREAD_NUM) //ֻ��0~1023���߳�
	{
		//ÿ���̼߳���1536��Ԫ��
		for(unsigned int i = tid; i < DATA_SIZE; i = i + THREAD_NUM)
			sum[i] = (a[i] - b[i]) * (a[i] - b[i]);
	}
}

int main()
{
	//�������϶�����������ʼ��Ϊ0
	int *a = new int[DATA_SIZE];
	memset(a, 0, sizeof(int) * DATA_SIZE);
	int *b = new int[DATA_SIZE];
	memset(b, 0, sizeof(int) * DATA_SIZE);
	int *sum = new int[DATA_SIZE];
	memset(sum, 0, sizeof(int) * DATA_SIZE);

	//�����������
	srand(time(NULL));
	genVector(a);
	genVector(b);

	//�����������϶�Ӧ���豸����
	int *dev_a, *dev_b, *dev_sum;
	//���豸�Ϸ���洢�ռ�
	HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int) * DATA_SIZE)); 
	HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int) * DATA_SIZE));
	HANDLE_ERROR(hipMalloc((void**)&dev_sum, sizeof(int) * DATA_SIZE));
	//���������豸��
	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice));

	//�����豸�ϵ�ִ��ʱ��
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//��ʱ��ʼ
	hipEventRecord(start, 0); 
	//���豸�Ͻ��м���
	vectorDistance<<<1, THREAD_NUM>>>(dev_a, dev_b, dev_sum);
	//��ʱ����
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	//��ȡ��¼��ʱ��
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	cout << "GPU time: " << elapsedTime << endl;

	//�Ѽ��������豸����������
	HANDLE_ERROR(hipMemcpy(sum, dev_sum, sizeof(int) * DATA_SIZE, hipMemcpyDeviceToHost));
	//�ͷ��豸�洢�ռ�
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_sum));

	int result = 0;
	for(int i = 0; i < DATA_SIZE; i++)
		result += sum[i];

	//�������
	float distance = sqrt((float)result);
	cout << "(GPU)distance = " << distance << endl;

	float d = 0.0;
	clock_t sTime = clock();
	d = dis(a, b);
	clock_t eTime = clock();
	cout << "CPU time: " << eTime - sTime << endl;
	cout << "(CPU)distance = " << d << endl;

	//�ͷ������ϵĴ洢�ռ�
	delete [] a;
	delete [] b;
	delete [] sum;
	system("pause");
    return 0;
}
