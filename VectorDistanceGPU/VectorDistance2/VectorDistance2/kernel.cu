#include "hip/hip_runtime.h"
/**
	���block���в���
	һ�η���32*1024���߳�
	ѭ��48��
**/

#include "hip/hip_runtime.h"
#include ""
#include "book.h"
#include <iostream>
#include <math.h>
#include <time.h>

using namespace std;

#define BLOCK_NUM 32
#define	THREAD_NUM 1024
#define DATA_SIZE 1572864

void genVector(int *a)
{
	for(int i = 0; i < DATA_SIZE; i++)
		a[i] = rand() % 10;
}

float dis(int *a, int *b)
{
	int sum = 0.0;
	float d = 0.0;
	for(int i = 0; i < DATA_SIZE; i++)
	{
		float temp = a[i] - b[i];
		sum += temp * temp;
	}
	d = sqrt((float)sum);
	return d;
}

__global__ void vectorDistance(int *a, int *b, int *sum)
{
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//һ�η���32*1024���߳�
	if(tid < BLOCK_NUM * THREAD_NUM)
	{
		for(unsigned int i = tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM)
			sum[i] = (a[i] - b[i]) * (a[i] - b[i]);
	}
}

int main()
{
	//�����϶�������
	int *a = new int[DATA_SIZE];
	memset(a, 0, sizeof(int) * DATA_SIZE);
	int *b = new int[DATA_SIZE];
	memset(b, 0, sizeof(int) * DATA_SIZE);
	int *sum = new int[DATA_SIZE];
	memset(sum, 0, sizeof(int) * DATA_SIZE);

	//�����������
	srand(time(NULL));
	genVector(a);
	genVector(b);

	//�豸�϶�����Ӧ����
	int *dev_a, *dev_b, *dev_sum;
	//���豸�Ͽ��ٴ洢�ռ�
	HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int) * DATA_SIZE));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int) * DATA_SIZE));
	HANDLE_ERROR(hipMalloc((void**)&dev_sum, sizeof(int) * DATA_SIZE));

	//�������ݵ��豸��
	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice));
	
	//�����豸��ִ��ʱ��
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//��ʱ��ʼ
	hipEventRecord(start, 0);
	//���豸�Ͻ��м���
	vectorDistance<<<BLOCK_NUM, THREAD_NUM>>>(dev_a, dev_b, dev_sum);
	//��ʱ����
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	//����GPU��ִ��ʱ��
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "GPU time: " << elapsedTime << endl;

	//�����������
	HANDLE_ERROR(hipMemcpy(sum, dev_sum, sizeof(int) * DATA_SIZE, hipMemcpyDeviceToHost));
	//�ͷ��豸�洢�ռ�
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_sum));

	int result = 0;
	for(int i = 0; i < DATA_SIZE; i++)
		result += sum[i];

	//�������
	float distance = sqrt((float)result);
	cout << "(GPU)distance = " << distance << endl;

	float d = 0.0;
	clock_t sTime = clock();
	d = dis(a, b);
	clock_t eTime = clock();
	cout << "CPU time: " << eTime - sTime << endl;
	cout << "(CPU)distance = " << d << endl;

	//�ͷ������ϵĴ洢�ռ�
	delete [] a;
	delete [] b;
	delete [] sum;
	system("pause");
    return 0;
}

