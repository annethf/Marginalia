#include "hip/hip_runtime.h"
/**
	ʹ�ù���洢��
	һ�η���32*1024���̣߳�ѭ��32��
**/

#ifndef __HIPCC__  
    #define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "book.h"
#include <iostream>
#include <math.h>
#include <time.h>

using namespace std;

#define BLOCK_NUM 32
#define	THREAD_NUM 1024
#define DATA_SIZE 1572864

void genVector(int *a)
{
	for(int i = 0; i < DATA_SIZE; i++)
		a[i] = rand() % 10;
}

float dis(int *a, int *b)
{
	int sum = 0;
	float d = 0.0;
	for(int i = 0; i < DATA_SIZE; i++)
	{
		int temp = a[i] - b[i];
		sum += temp * temp;
	}
	d = sqrt((float)sum);
	return d;
}

__global__ void vectorDistance(int *a, int *b, int *sum)
{
	extern __shared__ int shared[];
	unsigned int tid = threadIdx.x;
	shared[tid] = 0;
	//һ�η���32*1024���߳�
	if(tid < BLOCK_NUM * THREAD_NUM)
	{
		for(unsigned int i = blockIdx.x * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM)
			shared[tid] += (a[i] - b[i]) * (a[i] - b[i]);
		//�߳�ͬ��
		__syncthreads();
	}
	if(tid == 0)
	{
		for(int i = 1; i < THREAD_NUM; i++)
			shared[0] += shared[i];
		sum[blockIdx.x] = shared[0];
	}
}

int main()
{
	//�����϶�������
	int *a = new int[DATA_SIZE];
	memset(a, 0, sizeof(int) * DATA_SIZE);
	int *b = new int[DATA_SIZE];
	memset(b, 0, sizeof(int) * DATA_SIZE);
	int *sum = new int[BLOCK_NUM];
	memset(sum, 0, sizeof(int) * BLOCK_NUM);

	//�����������
	srand(time(NULL));
	genVector(a);
	genVector(b);

	//�豸�϶�����Ӧ����
	int *dev_a, *dev_b, *dev_sum;
	//���豸�Ͽ��ٴ洢�ռ�
	HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int) * DATA_SIZE));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int) * DATA_SIZE));
	HANDLE_ERROR(hipMalloc((void**)&dev_sum, sizeof(int) * BLOCK_NUM));

	//�������ݵ��豸��
	HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice));
	
	//�����豸��ִ��ʱ��
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//��ʱ��ʼ
	hipEventRecord(start, 0);
	//���豸�Ͻ��м���
	vectorDistance<<<BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int)>>>(dev_a, dev_b, dev_sum);
	//��ʱ����
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	//����GPU��ִ��ʱ��
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "GPU time: " << elapsedTime << endl;
	//�����������
	HANDLE_ERROR(hipMemcpy(sum, dev_sum, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost));
	//�ͷ��豸�洢�ռ�
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_sum));

	int result = 0;
	for(int i = 0; i < BLOCK_NUM; i++)
		result += sum[i];

	//�������
	float distance = sqrt((float)result);
	cout << "(GPU)distance = " << distance << endl;

	float d = 0.0;
	clock_t sTime = clock();
	d = dis(a, b);
	clock_t eTime = clock();
	cout << "CPU time: " << eTime - sTime << endl;
	cout << "(CPU)distance = " << d << endl;

	//�ͷ������ϵĴ洢�ռ�
	delete [] a;
	delete [] b;
	delete [] sum;
	system("pause");
    return 0;
}